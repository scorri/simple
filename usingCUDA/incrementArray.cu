/*
Simple non visual example to attempt creating a makefile for cuda
This example will increment the contents of an array by 1
This program accepts command line argument to set the 
size of the array and the block size otherwise it uses default values.
No validation checks on these inputs for non integer args or 
if they exceed the maximum block size for device
*/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sstream>

// increment array on Host
void hostIncrement(int* a, int N)
{
	for(int i=0;i<N;i++)
	{
		a[i]=a[i]+1;
	}

	return;
}

// increment array on Device
__global__ void deviceIncrement(int* a, int N)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(idx <N)
	{
		a[idx] = a[idx]+1;
	}
}

// main function
int main(int argc, char** argv)
{
	int N;
	int blockSize;
	// Check command line arguments
	if(argc == 1)
	{
		// none given, use default;
		N=16;
		blockSize=4;
	}
	else
	{
		std::istringstream(argv[1]) >> N;	
		std::istringstream(argv[2]) >> blockSize;
	}

	// calculate size of arrays
	size_t size=N*sizeof(int);

	// Allocate memory for arrays on host	
	int* a_h = (int*)malloc(size);
	int* b_h = (int*)malloc(size);

	// Allocate memory on device
	int* a_d;	
	hipMalloc((void**) &a_d, size);

	// Initialise array data
	for(int i=0;i<N;i++)
	{
		a_h[i] = i;
	}

	// Copy data from host to device
	hipMemcpy(a_d, a_h, sizeof(int)*N, hipMemcpyHostToDevice);
	
	// Do calculation on host
	hostIncrement(a_h, N);

	// Do calculation on device
	int nBlocks = N/blockSize + (N%blockSize == 0?0:1);
	
	deviceIncrement <<< nBlocks, blockSize >>> (a_d, N);

	// Retrieve results from device and store in b_h
	hipMemcpy(b_h, a_d, sizeof(int)*N, hipMemcpyDeviceToHost);

	for(int i=0;i<N;i++)
	{
		assert(a_h[i] == b_h[i]);
		printf("host - %d, device - %d\n", a_h[i], b_h[i]);
	}

	// Release memory
	free(a_h);
	free(b_h);
	hipFree(a_d);
	
	return 0;
}
